#include "hip/hip_runtime.h"
#define CUDACCs
#include"hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include""

texture<float, 1, hipReadModeElementType> gridData1D;
texture<float, 1, hipReadModeElementType> sumData2D;
texture<float, 1, hipReadModeElementType> squmData2D;
texture<float, 1, hipReadModeElementType> imageData2D;
texture<float, 1, hipReadModeElementType> features2D;
texture<float, 1, hipReadModeElementType> totalFeatures1D;
texture<float, 1, hipReadModeElementType>  poster2D;

void setGird(float  sgrid[], int gridLength) {
	float *dev_grid;
	int grid_data_size = sizeof(float)*gridLength;
	hipMalloc((void**)&dev_grid, grid_data_size);
	hipMemcpy(dev_grid, sgrid, grid_data_size, hipMemcpyHostToDevice);
	hipBindTexture(0, gridData1D, dev_grid);
}

void setSumAndSQum(int w, int h, int squm[], float sum[], int squmLen, int sumLen) {
	hipArray *sumArray;
	hipArray *squmArray;
	int squm_data_size = sizeof(float)*squmLen;
	int sum_data_size = sizeof(float)*sumLen;

	hipChannelFormatDesc chDesc6 = hipCreateChannelDesc<int>();
	hipChannelFormatDesc  chDesc7 = hipCreateChannelDesc<float>();
	hipMallocArray(&sumArray, &chDesc6, w, h);
	hipMallocArray(&squmArray, &chDesc7, w, h);

	hipMemcpyToArray(sumArray, 0, 0, sum, sum_data_size, hipMemcpyHostToDevice);
	hipMemcpyToArray(squmArray, 0, 0, squm, squm_data_size, hipMemcpyHostToDevice);
	hipBindTextureToArray(sumData2D, sumArray);
	hipBindTextureToArray(squmData2D, squmArray);
}
// Ч������ ������ʹ��
__global__ void varClassifier(float *tfans, int gird_w, float var) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= gird_w) {
		return;
	}
	int index = tid;
	int box_x = tex1Dfetch(gridData1D, index); //ʹ�������ڴ�
	int box_y = tex1Dfetch(gridData1D, index + gird_w);
	int box_w = tex1Dfetch(gridData1D, index + gird_w * 2);
	int box_h = tex1Dfetch(gridData1D, index + gird_w * 3);
	int scale_idx = tex1Dfetch(gridData1D, index + gird_w * 4);

	float brs = tex2D(sumData2D, box_x + box_w, box_y + box_h);
	float bls = tex2D(sumData2D, box_y + box_h);
	float trs = tex2D(sumData2D, box_x + box_w, box_y);
	float tls = tex2D(sumData2D, box_x, box_y);

	float brsq = tex2D(squmData2D, box_x + box_w, box_y + box_h);
	float blsq = tex2D(squmData2D, box_x, box_y + box_h);
	float trsq = tex2D(squmData2D, box_x + box_w, box_y);
	float tlsq = tex2D(squmData2D, box_x, box_y);

	float mean = (brs + tls - trs - bls) / ((float)box_w*box_h);
	float sqmean = (brsq + tlsq - trsq - blsq) / ((float)box_w*box_h);
	float temp = sqmean - mean*mean;//return   sqmean-mean*mean;

	if (temp >= var) {
		tfans[tid] = tid;
	}
	else {
		tfans[tid] = -1;
	}

}

__global__ void upPoker(float *dev_poater, int *dev_upPosInd, float * dev_upPos, int pos_pitch)
{
	int tid = threadIdx.x;
	int idx = dev_upPos[tid];
	float var = dev_upPos[tid];
	*((float*)((char*)dev_poater + pos_pitch *tid) + idx) = var;
}

__global__ void collectionClassifier(int *ans, int h, float threhold, int grid_w, int nstructs, int structSize)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= h) return;

	int index = tex1Dfetch(totalFeatures1D, tid);
	int box_x = tex1Dfetch(gridData1D, tid); //ʹ�������ڴ�
	int box_y = tex1Dfetch(gridData1D, index + grid_w);
	int box_w = tex1Dfetch(gridData1D, index + grid_w * 2);
	int box_h = tex1Dfetch(gridData1D, index + grid_w * 3);
	int scale_idx = tex1Dfetch(gridData1D, index + grid_w * 4);
	int leaf, x1, x2, y1, y2, imbig = 0;
	float votes = 0;
	float point1, point2;

	for (int t = 0; t < nstructs; t++) {
		leaf = 0;

		for (int f = 0; f < structSize; f++) {
			x1 = tex2D(features2D, t*structSize + f, scale_idx);
			x2 = tex2D(features2D, t*structSize + structSize *nstructs * 2 + f, scale_idx);
			y1 = tex2D(features2D, t*structSize + structSize *nstructs + f, scale_idx);
			y2 = tex2D(features2D, t*structSize + structSize *nstructs * 3 + f, scale_idx);

			point1 = tex2D(imageData2D, box_x + x1, box_y + y1);
			point2 = tex2D(imageData2D, box_x + x2, box_y + y2);

			if (point1 > point2) {
				imbig = 1;
			}
			else {
				imbig = 0;
			}
			leaf = (leaf << 1) + imbig;
		}
		ans[tid*(nstructs + 2) + t] = leaf;
		votes = tex2D(poster2D, leaf, t);
	}

	float conf = votes;
	ans[tid*(nstructs + 2) + nstructs] = conf;

	if (conf >threhold) {
		ans[tid*(nstructs + 2) + nstructs + 1] = index;
	}
	else {
		ans[tid*(nstructs + 2) + nstructs + 1] = -1;
	}

	/*

	*/
	int* runCollectionClassifier(int varisNum,  int ansLength,int h, float threhold, int grid_w, int nstructs, int structSize) {
		
		int * inAns;
		int *outAns;
		outAns = new int [ansLength];
		hipMalloc((void**)&inAns, ansLength);
		dim3 block(varisNum,1,1);
		dim3 grid(10, 13,1);
		collectionClassifier << <grid, block, 0 >> > (inAns, h, threhold, grid_w, nstructs, structSize);
		hipMemcpy(outAns, inAns, ansLength, hipMemcpyDeviceToHost);
		return outAns��
	}
}
